#include "hip/hip_runtime.h"
#pragma once

#include "shared.h"

using namespace Shared;

extern "C" __constant__ PipelineLaunchParameters plp;

extern "C" __global__
void __raygen__fill() {
    uint3 launchIndex = optixGetLaunchIndex();
    RayGenData* rgData = (RayGenData*)optixGetSbtDataPointer();
    int32_t index = plp.imageSize.x * launchIndex.y + launchIndex.x;
    float x = (float)launchIndex.x / plp.imageSize.x;
    float y = (float)launchIndex.y / plp.imageSize.y;
    plp.outputBuffer[index] = make_float4(rgData->r * x, rgData->g * y, rgData->b, 1.0f);
}
