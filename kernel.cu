#include "hip/hip_runtime.h"
#pragma once

#include "shared.h"

#define RT_FUNCTION __forceinline__ __device__
#define RT_PROGRAM extern "C" __global__

namespace Sample {

using namespace Shared;

RT_FUNCTION float3 normalize(const float3 &v) {
    float rl = 1.0f / norm3df(v.x, v.y, v.z);
    return make_float3(v.x * rl, v.y * rl, v.z * rl);
}

RT_FUNCTION float4 make_float4(const float3 &v, float w) {
    return ::make_float4(v.x, v.y, v.z, w);
}



extern "C" __constant__ PipelineLaunchParameters plp;



// JP: ���̃N���X�̂悤�ɃV�X�e�}�e�B�b�N��uint32_t&�ɂ����ɁA
//     �ʂɓK�؂ȃy�C���[�h�̓n�������l�����ق������\�͗ǂ���������Ȃ��B
template <typename PayloadType>
union PayloadAccessor {
    PayloadType raw;
    uint32_t asUInt[(sizeof(PayloadType) + 3) / 4];

    RT_FUNCTION PayloadAccessor() {
        for (int i = 0; i < sizeof(asUInt) / 4; ++i)
            asUInt[i] = optixUndefinedValue();
    }

    RT_FUNCTION uint32_t &operator[](uint32_t idx) {
        return asUInt[idx];
    }

    RT_FUNCTION void getAll() {
        constexpr uint32_t numSlots = sizeof(asUInt) / 4;
        if (numSlots > 0)
            asUInt[0] = optixGetPayload_0();
        if (numSlots > 1)
            asUInt[1] = optixGetPayload_1();
        if (numSlots > 2)
            asUInt[2] = optixGetPayload_2();
        if (numSlots > 3)
            asUInt[3] = optixGetPayload_3();
        if (numSlots > 4)
            asUInt[4] = optixGetPayload_4();
        if (numSlots > 5)
            asUInt[5] = optixGetPayload_5();
        if (numSlots > 6)
            asUInt[6] = optixGetPayload_6();
        if (numSlots > 7)
            asUInt[7] = optixGetPayload_7();
    }

    RT_FUNCTION void setAll() const {
        constexpr uint32_t numSlots = sizeof(asUInt) / 4;
        if (numSlots > 0)
            optixSetPayload_0(asUInt[0]);
        if (numSlots > 1)
            optixSetPayload_1(asUInt[1]);
        if (numSlots > 2)
            optixSetPayload_2(asUInt[2]);
        if (numSlots > 3)
            optixSetPayload_3(asUInt[3]);
        if (numSlots > 4)
            optixSetPayload_4(asUInt[4]);
        if (numSlots > 5)
            optixSetPayload_5(asUInt[5]);
        if (numSlots > 6)
            optixSetPayload_6(asUInt[6]);
        if (numSlots > 7)
            optixSetPayload_7(asUInt[7]);
    }
};



struct Ray {
    float3 origin;
    float3 direction;
    float tmin;
    float tmax;
    float time;

    RT_FUNCTION static Ray getWorld() {
        Ray ret;
        ret.origin = optixGetWorldRayOrigin();
        ret.direction = optixGetWorldRayDirection();
        ret.tmin = optixGetRayTmin();
        ret.tmax = optixGetRayTmax();
        ret.time = optixGetRayTime();
        return ret;
    }
    RT_FUNCTION static Ray getObject() {
        Ray ret;
        ret.origin = optixGetObjectRayOrigin();
        ret.direction = optixGetObjectRayDirection();
        ret.tmin = optixGetRayTmin();
        ret.tmax = optixGetRayTmax();
        ret.time = optixGetRayTime();
        return ret;
    }
};

struct SearchRayPayload {
    float3 contribution;
};

struct VisibilityRayPayload {
    float visibility;
};

struct HitPointParameter {
    float b0, b1;
    int32_t primIndex;

    RT_FUNCTION static HitPointParameter get() {
        HitPointParameter ret;
        if (optixIsTriangleHit()) {
            float2 bc = optixGetTriangleBarycentrics();
            ret.b0 = 1 - bc.x - bc.y;
            ret.b1 = bc.x;
        }
        else {
            ret.b0 = __uint_as_float(optixGetAttribute_0());
            ret.b1 = __uint_as_float(optixGetAttribute_1());
        }
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};



RT_PROGRAM void __raygen__fill() {
    uint3 launchIndex = optixGetLaunchIndex();
    int32_t index = plp.imageSize.x * launchIndex.y + launchIndex.x;

    float x = (float)(launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = (float)(launchIndex.y + 0.5f) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = make_float3(0, 0, 3);
    float3 direction = normalize(make_float3(vw * (x - 0.5f), vh * (0.5f - y), -1));

    PayloadAccessor<SearchRayPayload> payload;
    optixTrace(plp.topGroup, origin, direction, 0.0f, INFINITY, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
               RayType_Search, NumRayTypes, RayType_Search,
               payload[0], payload[1], payload[2]);

    plp.outputBuffer[index] = make_float4(payload.raw.contribution, 1.0f);
}

RT_PROGRAM void __miss__searchRay() {
    PayloadAccessor<SearchRayPayload> payload;

    payload.raw.contribution = make_float3(0.0f, 0.0f, 0.05f);
    
    payload.setAll();
}

RT_PROGRAM void __closesthit__shading() {
    const auto &sbtrData = *(HitGroupData*)optixGetSbtDataPointer();

    auto hitPointParam = HitPointParameter::get();

    PayloadAccessor<SearchRayPayload> payload;
    payload.raw.contribution = sbtrData.mat.albedo;

    payload.setAll();
}

RT_PROGRAM void __anyhit__visibility() {
    PayloadAccessor<VisibilityRayPayload> payload;

    payload.raw.visibility = 0.0f;
    payload.setAll();

    optixTerminateRay();
}

}
