#include "micro_map_generator_private.h"
#if !defined(__INTELLISENSE__)
#include <hipcub/hipcub.hpp>
#endif

struct DirectedEdgeLessOp {
    CUDA_DEVICE_FUNCTION CUDA_INLINE bool operator()(
        const shared::DirectedEdge &l, const shared::DirectedEdge &r) const {
        return l < r;
    }
};

size_t __getScratchMemSizeForSortDirectedEdges(uint32_t numEdges) {
    size_t size;
    hipcub::DeviceMergeSort::StableSortPairs<shared::DirectedEdge*, uint32_t*>(
        nullptr, size,
        nullptr, nullptr, numEdges, DirectedEdgeLessOp());
    return size;
}

// TODO: Use radix sort?
void __sortDirectedEdges(
    shared::DirectedEdge* edges, uint32_t* edgeIndices, uint32_t numEdges,
    void* scratchMem, size_t scratchMemSize) {
    hipcub::DeviceMergeSort::StableSortPairs<shared::DirectedEdge*, uint32_t*>(
        scratchMem, scratchMemSize,
        edges, edgeIndices, numEdges, DirectedEdgeLessOp());
}



struct TriTexCoordTupleLessOp {
    CUDA_DEVICE_FUNCTION CUDA_INLINE bool operator()(
        const shared::TriTexCoordTuple &l, const shared::TriTexCoordTuple &r) const {
        return l < r;
    }
};

size_t __getScratchMemSizeForSortTriTexCoordTuples(uint32_t numTriangles) {
    size_t size;
    hipcub::DeviceMergeSort::StableSortPairs<shared::TriTexCoordTuple*, uint32_t*>(
        nullptr, size,
        nullptr, nullptr, numTriangles, TriTexCoordTupleLessOp());
    return size;
}

// TODO: Use radix sort?
void __sortTriTexCoordTuples(
    shared::TriTexCoordTuple* tuples, uint32_t* triIndices, uint32_t numTriangles,
    void* scratchMem, size_t scratchMemSize) {
    hipcub::DeviceMergeSort::StableSortPairs<shared::TriTexCoordTuple*, uint32_t*>(
        scratchMem, scratchMemSize,
        tuples, triIndices, numTriangles, TriTexCoordTupleLessOp());
}
