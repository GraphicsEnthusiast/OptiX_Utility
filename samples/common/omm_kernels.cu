#include "hip/hip_runtime.h"
﻿#include "omm_generator.h"
#include "optix_micromap.h"

using namespace shared;

static constexpr uint32_t WarpSize = 32;

struct Triangle {
    uint32_t index0;
    uint32_t index1;
    uint32_t index2;
};

CUDA_DEVICE_FUNCTION CUDA_INLINE float fetchAlpha(
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    int2 pix) {
    union Alias {
        float4 f4;
        float2 f2;
        float f;
        float a[4];
        CUDA_DEVICE_FUNCTION Alias() {}
    } alias;

    float2 texCoord = make_float2(
        (pix.x + 0.5f) / texSize.x,
        (pix.y + 0.5f) / texSize.y);

    float alpha;
    if (numChannels == 4) {
        alias.f4 = tex2DLod<float4>(texture, texCoord.x, texCoord.y, 0.0f);
        alpha = alias.a[alphaChannelIdx];
    }
    else if (numChannels == 2) {
        alias.f2 = tex2DLod<float2>(texture, texCoord.x, texCoord.y, 0.0f);
        alpha = alias.a[alphaChannelIdx];
    }
    else {
        alpha = tex2DLod<float>(texture, texCoord.x, texCoord.y, 0.0f);
    }
    return alpha;
}

CUDA_DEVICE_FUNCTION CUDA_INLINE bool isTransparent(float alpha) {
    return alpha < 0.5f;
}



enum class FlatTriangleType {
    BottomFlat,
    TopFlat,
};

template <FlatTriangleType triType, bool ignoreFirstLine>
CUDA_DEVICE_FUNCTION CUDA_INLINE void rasterizeFlatTriangle(
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    const float2 ps[2],
    uint32_t* numTransparentPixels, uint32_t* numPixels) {
    float invBeginSlope;
    float invEndSlope;
    float curXFBegin;
    float curXFEnd;
    int32_t curY;
    int32_t yEnd;
    if constexpr (triType == FlatTriangleType::BottomFlat) {
        invBeginSlope = (ps[1].x - ps[0].x) / (ps[1].y - ps[0].y);
        invEndSlope = (ps[2].x - ps[0].x) / (ps[2].y - ps[0].y);

        curXFBegin = ps[0].x;
        curXFEnd = ps[0].x;
        curY = static_cast<int32_t>(ps[0].y);
        yEnd = static_cast<int32_t>(ps[1].y);
    }
    else /*if constexpr (triType == FlatTriangleType::TopFlat)*/ {
        invEndSlope = -(ps[2].x - ps[0].x) / (ps[2].y - ps[0].y);
        invBeginSlope = -(ps[2].x - ps[1].x) / (ps[2].y - ps[1].y);

        curXFBegin = ps[2].x;
        curXFEnd = ps[2].x;
        curY = static_cast<int32_t>(ps[2].y);
        yEnd = static_cast<int32_t>(ps[0].y) + ignoreFirstLine;
    }

    int32_t curX = static_cast<int32_t>(curXFBegin);
    int32_t curXEnd = static_cast<int32_t>(curXFEnd);
    uint32_t curNumItemsPerWarp = 0;
    int2 item = make_int2(INT32_MAX, INT32_MAX);
    while (triType == FlatTriangleType::BottomFlat ? curY <= yEnd : curY >= yEnd) {
        const uint32_t numItemsToFill =
            min(curXEnd - curX + 1, static_cast<int32_t>(WarpSize - curNumItemsPerWarp));
        if (threadIdx.x >= curNumItemsPerWarp &&
            threadIdx.x < (curNumItemsPerWarp + numItemsToFill))
            item = make_int2(curX + threadIdx.x - curNumItemsPerWarp, curY);
        curNumItemsPerWarp += numItemsToFill;
        *numPixels += numItemsToFill;
        if (curNumItemsPerWarp == WarpSize) {
            const float alpha = fetchAlpha(texture, texSize, numChannels, alphaChannelIdx, item);
            const uint32_t numTrsInWarp = __popc(__ballot_sync(0xFFFFFFFF, isTransparent(alpha)));
            *numTransparentPixels += numTrsInWarp;
            curNumItemsPerWarp = 0;
        }
        curX += numItemsToFill;
        if (curX > curXEnd) {
            curXFBegin += invBeginSlope;
            curXFEnd += invEndSlope;
            curX = static_cast<int32_t>(curXFBegin);
            curXEnd = static_cast<int32_t>(curXFEnd);
            if constexpr (triType == FlatTriangleType::BottomFlat)
                ++curY;
            else /*if constexpr (triType == FlatTriangleType::TopFlat)*/
                --curY;
        }
    }
    if (curNumItemsPerWarp > 0) {
        float alpha = 1.0f;
        if (threadIdx.x < curNumItemsPerWarp)
            alpha = fetchAlpha(texture, texSize, numChannels, alphaChannelIdx, item);
        const uint32_t numTrsInWarp = __popc(__ballot_sync(0xFFFFFFFF, isTransparent(alpha)));
        *numTransparentPixels += numTrsInWarp;
    }
}

CUDA_DEVICE_FUNCTION CUDA_INLINE void evaluateSingleTriangleTransparency(
    uint32_t triIdx,
    const uint8_t* texCoords, uint64_t vertexStride,
    const uint8_t* triangles, uint64_t triangleStride, uint32_t numTriangles,
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    uint32_t* numTransparentPixels, uint32_t* numPixels) {
    auto &tri = reinterpret_cast<const Triangle &>(triangles[triangleStride * triIdx]);
    const float2 tcs[] = {
        reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index0]),
        reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index1]),
        reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index2]),
    };
    float2 fPixs[3] = {
        make_float2(texSize.x * tcs[0].x, texSize.y * tcs[0].y),
        make_float2(texSize.x * tcs[1].x, texSize.y * tcs[1].y),
        make_float2(texSize.x * tcs[2].x, texSize.y * tcs[2].y)
    };

    const auto swap = [](float2 &a, float2 &b) {
        float2 temp = a;
        a = b;
        b = temp;
    };

    // Sort vertices to be Y-ascending
    if (fPixs[0].y > fPixs[1].y)
        swap(fPixs[0], fPixs[1]);
    if (fPixs[1].y > fPixs[2].y)
        swap(fPixs[1], fPixs[2]);
    if (fPixs[0].y > fPixs[1].y)
        swap(fPixs[0], fPixs[1]);
    
    *numTransparentPixels = 0;
    *numPixels = 0;

    // Top-Flat
    if (fPixs[0].y == fPixs[1].y) {
        // Make the triangle CCW
        if (fPixs[0].x < fPixs[1].x)
            swap(fPixs[0], fPixs[1]);

        rasterizeFlatTriangle<FlatTriangleType::TopFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            fPixs,
            numTransparentPixels, numPixels);
    }
    // Bottom-Flat
    else if (fPixs[1].y == fPixs[2].y) {
        // Make the triangle CCW
        if (fPixs[1].x >= fPixs[2].x)
            swap(fPixs[1], fPixs[2]);

        rasterizeFlatTriangle<FlatTriangleType::BottomFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            fPixs,
            numTransparentPixels, numPixels);
    }
    // General
    else {
        const float t = (fPixs[1].y - fPixs[0].y) / (fPixs[2].y - fPixs[0].y);
        const float2 newP = make_float2(
            fPixs[0].x + t * (fPixs[2].x - fPixs[0].x),
            fPixs[1].y);

        float2 ps[3];
        ps[0] = fPixs[0];
        ps[1] = fPixs[1];
        ps[2] = newP;
        // Make the triangle CCW
        if (ps[1].x >= ps[2].x)
            swap(ps[1], ps[2]);

        rasterizeFlatTriangle<FlatTriangleType::BottomFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            numTransparentPixels, numPixels);

        ps[0] = newP;
        ps[1] = fPixs[1];
        ps[2] = fPixs[2];
        // Make the triangle CCW
        if (ps[0].x < ps[1].x)
            swap(ps[0], ps[1]);

        rasterizeFlatTriangle<FlatTriangleType::TopFlat, true>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            numTransparentPixels, numPixels);
    }
}

CUDA_DEVICE_KERNEL void countOMMFormats(
    const uint8_t* texCoords, uint64_t vertexStride,
    const uint8_t* triangles, uint64_t triangleStride, uint32_t numTriangles,
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    OMMFormat maxSubdivLevel, int32_t subdivLevelBias,
    volatile uint32_t* numFetchedTriangles,
    uint32_t* ommFormatCounts, uint64_t* ommSizes) {
    while (true) {
        uint32_t curNumFetches;
        if (threadIdx.x == 0)
            curNumFetches = *numFetchedTriangles;
        curNumFetches = __shfl_sync(0xFFFFFFFF, curNumFetches, 0);
        if (curNumFetches >= numTriangles)
            return;

        uint32_t baseTriIdx;
        if (threadIdx.x == 0)
            baseTriIdx = atomicAdd(const_cast<uint32_t*>(numFetchedTriangles), WarpSize);
        baseTriIdx = __shfl_sync(0xFFFFFFFF, baseTriIdx, 0);

        for (uint32_t triSubIdx = 0; triSubIdx < WarpSize; ++triSubIdx) {
            // JP: Warp中の全スレッドが同じ三角形を処理する。
            // EN: All the threads in a warp process the same triangle.
            const uint32_t triIdx = baseTriIdx + triSubIdx;
            if (triIdx >= numTriangles)
                return;

            uint32_t numTransparentPixels;
            uint32_t numPixels;
            evaluateSingleTriangleTransparency(
                triIdx,
                texCoords, vertexStride, triangles, triangleStride, numTriangles,
                texture, texSize, numChannels, alphaChannelIdx,
                &numTransparentPixels, &numPixels);

            if (threadIdx.x == 0) {
                const bool singleState = numTransparentPixels == 0 || numTransparentPixels == numPixels;
                constexpr int32_t minLevel = OMMFormat_None;
                const int32_t maxLevel = static_cast<int32_t>(maxSubdivLevel);
                const int32_t level = singleState ? 0 :
                    min(max(static_cast<int32_t>(
                        std::log(static_cast<float>(numPixels)) / std::log(4.0f)
                        ) - 4 + subdivLevelBias, minLevel), maxLevel); // -4: ad-hoc offset
                atomicAdd(&ommFormatCounts[level], 1u);

                const uint32_t ommSizeInBits = level == 0 ? 0 : 2 * (1 << (2 * level));
                const uint32_t ommSizeInBytes = (ommSizeInBits + 7) / 8;
                ommSizes[triIdx] = ommSizeInBytes;
            }
        }
    }
}



CUDA_DEVICE_KERNEL void createOMMDescriptors(
    const uint64_t* ommOffsets, uint32_t numTriangles,
    uint32_t* descCounter,
    OptixOpacityMicromapDesc* ommDescs, void* ommIndices, uint32_t ommIndexSize) {
    const uint32_t triIdx = blockDim.x * blockIdx.x + threadIdx.x;
    if (triIdx >= numTriangles)
        return;

    const auto ommIndices8 = reinterpret_cast<int8_t*>(ommIndices);
    const auto ommIndices16 = reinterpret_cast<int16_t*>(ommIndices);
    const auto ommIndices32 = reinterpret_cast<int32_t*>(ommIndices);

    const uint64_t ommOffset = ommOffsets[triIdx];
    const uint32_t ommSize = static_cast<uint32_t>(ommOffsets[triIdx + 1] - ommOffset);
    const uint32_t numMicroTris = 4 * ommSize;
    const uint32_t ommLevel = tzcnt(numMicroTris) >> 1;
    if (ommLevel == 0) {
        const int32_t ommIndex = OPTIX_OPACITY_MICROMAP_PREDEFINED_INDEX_FULLY_OPAQUE;
        if (ommIndexSize == 1)
            ommIndices8[triIdx] = ommIndex;
        else if (ommIndexSize == 2)
            ommIndices16[triIdx] = ommIndex;
        else/* if (ommIndexSize == 4)*/
            ommIndices32[triIdx] = ommIndex;
        return;
    }

    const uint32_t descIdx = atomicAdd(descCounter, 1u);
    OptixOpacityMicromapDesc &ommDesc = ommDescs[descIdx];
    ommDesc.byteOffset = ommOffset;
    ommDesc.format = OPTIX_OPACITY_MICROMAP_FORMAT_4_STATE;
    ommDesc.subdivisionLevel = ommLevel;

    if (ommIndexSize == 1)
        ommIndices8[triIdx] = static_cast<int32_t>(descIdx);
    else if (ommIndexSize == 2)
        ommIndices16[triIdx] = static_cast<int32_t>(descIdx);
    else/* if (ommIndexSize == 4)*/
        ommIndices32[triIdx] = static_cast<int32_t>(descIdx);
}



template <FlatTriangleType triType, bool ignoreFirstLine>
CUDA_DEVICE_FUNCTION CUDA_INLINE void rasterizeFlatMicroTriangle(
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    const float2 ps[2],
    uint32_t* numTransparentPixels, uint32_t* numPixels) {
    float invBeginSlope;
    float invEndSlope;
    float curXFBegin;
    float curXFEnd;
    int32_t curY;
    int32_t yEnd;
    if constexpr (triType == FlatTriangleType::BottomFlat) {
        invBeginSlope = (ps[1].x - ps[0].x) / (ps[1].y - ps[0].y);
        invEndSlope = (ps[2].x - ps[0].x) / (ps[2].y - ps[0].y);

        curXFBegin = ps[0].x;
        curXFEnd = ps[0].x;
        curY = static_cast<int32_t>(ps[0].y);
        yEnd = static_cast<int32_t>(ps[1].y);
    }
    else /*if constexpr (triType == FlatTriangleType::TopFlat)*/ {
        invEndSlope = -(ps[2].x - ps[0].x) / (ps[2].y - ps[0].y);
        invBeginSlope = -(ps[2].x - ps[1].x) / (ps[2].y - ps[1].y);

        curXFBegin = ps[2].x;
        curXFEnd = ps[2].x;
        curY = static_cast<int32_t>(ps[2].y);
        yEnd = static_cast<int32_t>(ps[0].y) + ignoreFirstLine;
    }

    while (triType == FlatTriangleType::BottomFlat ? curY <= yEnd : curY >= yEnd) {
        int32_t curXBegin = static_cast<int32_t>(curXFBegin);
        int32_t curXEnd = static_cast<int32_t>(curXFEnd);
        for (int32_t curX = curXBegin; curX <= curXEnd; ++curX) {
            int2 item = make_int2(curX, curY);
            float alpha = fetchAlpha(texture, texSize, numChannels, alphaChannelIdx, item);
            if (isTransparent(alpha))
                ++*numTransparentPixels;
        }
        curXFBegin += invBeginSlope;
        curXFEnd += invEndSlope;
        if constexpr (triType == FlatTriangleType::BottomFlat)
            ++curY;
        else /*if constexpr (triType == FlatTriangleType::TopFlat)*/
            --curY;

        *numPixels += curXEnd - curXBegin + 1;
    }
}

CUDA_DEVICE_FUNCTION CUDA_INLINE uint32_t evaluateSingleMicroTriangle(
    float2 fPixA, float2 fPixB, float2 fPixC,
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx) {
    const auto swap = [](float2 &a, float2 &b) {
        float2 temp = a;
        a = b;
        b = temp;
    };

    // Sort vertices to be Y-ascending
    if (fPixA.y > fPixB.y)
        swap(fPixA, fPixB);
    if (fPixB.y > fPixC.y)
        swap(fPixB, fPixC);
    if (fPixA.y > fPixB.y)
        swap(fPixA, fPixB);

    float2 ps[3];
    ps[0] = fPixA;
    ps[1] = fPixB;
    ps[2] = fPixC;

    uint32_t numTrPixels = 0;
    uint32_t numPixels = 0;

    // Top-Flat
    if (ps[0].y == ps[1].y) {
        // Make the triangle CCW
        if (ps[0].x < ps[1].x)
            swap(ps[0], ps[1]);

        rasterizeFlatMicroTriangle<FlatTriangleType::TopFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            &numTrPixels, &numPixels);
    }
    // Bottom-Flat
    else if (ps[1].y == ps[2].y) {
        // Make the triangle CCW
        if (ps[1].x >= ps[2].x)
            swap(ps[1], ps[2]);

        rasterizeFlatMicroTriangle<FlatTriangleType::BottomFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            &numTrPixels, &numPixels);
    }
    // General
    else {
        const float t = (fPixB.y - fPixA.y) / (fPixC.y - fPixA.y);
        const float2 newP = make_float2(
            fPixA.x + t * (fPixC.x - fPixA.x),
            fPixB.y);

        ps[0] = fPixA;
        ps[1] = fPixB;
        ps[2] = newP;
        // Make the triangle CCW
        if (ps[1].x >= ps[2].x)
            swap(ps[1], ps[2]);

        rasterizeFlatMicroTriangle<FlatTriangleType::BottomFlat, false>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            &numTrPixels, &numPixels);

        ps[0] = newP;
        ps[1] = fPixB;
        ps[2] = fPixC;
        // Make the triangle CCW
        if (ps[0].x < ps[1].x)
            swap(ps[0], ps[1]);

        rasterizeFlatMicroTriangle<FlatTriangleType::TopFlat, true>(
            texture, texSize, numChannels, alphaChannelIdx,
            ps,
            &numTrPixels, &numPixels);
    }

    uint32_t state;
    if (numTrPixels == 0)
        state = OPTIX_OPACITY_MICROMAP_STATE_OPAQUE;
    else if (numTrPixels == numPixels)
        state = OPTIX_OPACITY_MICROMAP_STATE_TRANSPARENT;
    else if (2 * numTrPixels < numPixels)
        state = OPTIX_OPACITY_MICROMAP_STATE_UNKNOWN_OPAQUE;
    else
        state = OPTIX_OPACITY_MICROMAP_STATE_UNKNOWN_TRANSPARENT;

    return state;
}

CUDA_DEVICE_KERNEL void evaluateMicroTriangleTransparencies(
    const uint8_t* texCoords, uint64_t vertexStride,
    const uint8_t* triangles, uint64_t triangleStride, uint32_t numTriangles,
    hipTextureObject_t texture, uint2 texSize, uint32_t numChannels, uint32_t alphaChannelIdx,
    const uint64_t* ommOffsets,
    volatile uint32_t* numFetchedTriangles,
    uint8_t* opacityMicroMaps) {
    while (true) {
        if (*numFetchedTriangles >= numTriangles)
            return;

        uint32_t baseTriIdx;
        if (threadIdx.x == 0)
            baseTriIdx = atomicAdd(const_cast<uint32_t*>(numFetchedTriangles), WarpSize);
        baseTriIdx = __shfl_sync(0xFFFFFFFF, baseTriIdx, 0);

        for (uint32_t triSubIdx = 0; triSubIdx < WarpSize; ++triSubIdx) {
            // JP: Warp中の全スレッドが同じ三角形を処理する。
            // EN: All the threads in a warp process the same triangle.
            const uint32_t triIdx = baseTriIdx + triSubIdx;
            if (triIdx >= numTriangles)
                return;

            auto &tri = reinterpret_cast<const Triangle &>(triangles[triangleStride * triIdx]);
            const float2 tcA = reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index0]);
            const float2 tcB = reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index1]);
            const float2 tcC = reinterpret_cast<const float2 &>(texCoords[vertexStride * tri.index2]);
            const float2 fTexSize = make_float2(texSize.x, texSize.y);

            const uint64_t ommOffset = ommOffsets[triIdx];
            const uint32_t ommSize = static_cast<uint32_t>(ommOffsets[triIdx + 1] - ommOffset);
            uint8_t* const opacityMicroMap = opacityMicroMaps + ommOffset;

            // JP: Opacity Micro-Mapのクリア。
            // EN: Clear the opacity micro-map.
            const uint32_t numDwords = ommSize / 4;
            for (uint32_t dwBaseIdx = 0; dwBaseIdx < numDwords; dwBaseIdx += WarpSize) {
                const uint32_t dwIdx = dwBaseIdx + threadIdx.x;
                if (dwIdx < numDwords)
                    reinterpret_cast<uint32_t*>(opacityMicroMap)[dwIdx] = 0;
            }

            const uint32_t numMicroTris = 4 * ommSize;
            const uint32_t ommLevel = tzcnt(numMicroTris) >> 1;
            for (uint32_t microTriBaseIdx = 0; microTriBaseIdx < numMicroTris; microTriBaseIdx += WarpSize) {
                // JP: 各スレッドがマイクロ三角形のステートを計算する。
                // EN: Each thread computes the state of a micro triangle.
                // TODO: Upper/Lower Micro-Triangleを適切に振り分けてDivergenceを抑える。
                const uint32_t microTriIdx = microTriBaseIdx + threadIdx.x;
                if (microTriIdx >= numMicroTris)
                    break;

                float2 bc0, bc1, bc2;
                optixMicromapIndexToBaseBarycentrics(microTriIdx, ommLevel, bc0, bc1, bc2);

                const float2 fPix0 =
                    fTexSize * ((1.0f - (bc0.x + bc0.y)) * tcA + bc0.x * tcB + bc0.y * tcC);
                const float2 fPix1 =
                    fTexSize * ((1.0f - (bc1.x + bc1.y)) * tcA + bc1.x * tcB + bc1.y * tcC);
                const float2 fPix2 =
                    fTexSize * ((1.0f - (bc2.x + bc2.y)) * tcA + bc2.x * tcB + bc2.y * tcC);

                const uint32_t state = evaluateSingleMicroTriangle(
                    fPix0, fPix1, fPix2,
                    texture, texSize, numChannels, alphaChannelIdx);

                const uint32_t binIdx = microTriIdx / 16;
                const uint32_t offsetInBin = 2 * (microTriIdx % 16);
                atomicOr(
                    reinterpret_cast<uint32_t*>(opacityMicroMap) + binIdx,
                    state << offsetInBin);
            }
        }
    }
}