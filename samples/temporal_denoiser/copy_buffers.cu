#include "hip/hip_runtime.h"
#include "temporal_denoiser_shared.h"

using namespace Shared;

CUDA_DEVICE_KERNEL void copyToLinearBuffers(
    optixu::NativeBlockBuffer2D<float4> colorAccumBuffer,
    optixu::NativeBlockBuffer2D<float4> albedoAccumBuffer,
    optixu::NativeBlockBuffer2D<float4> normalAccumBuffer,
    float4* linearColorBuffer,
    float4* linearAlbedoBuffer,
    float4* linearNormalBuffer,
    uint2 imageSize) {
    uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                   blockDim.y * blockIdx.y + threadIdx.y);
    if (launchIndex.x >= imageSize.x ||
        launchIndex.y >= imageSize.y)
        return;

    uint32_t linearIndex = launchIndex.y * imageSize.x + launchIndex.x;
    linearColorBuffer[linearIndex] = colorAccumBuffer.read(launchIndex);
    linearAlbedoBuffer[linearIndex] = albedoAccumBuffer.read(launchIndex);
    float3 normal = getXYZ(normalAccumBuffer.read(launchIndex));
    if (normal.x != 0 || normal.y != 0 || normal.z != 0)
        normal = normalize(normal);
    linearNormalBuffer[linearIndex] = make_float4(normal, 1.0f);
}

CUDA_DEVICE_KERNEL void visualizeToOutputBuffer(
    void* linearBuffer,
    BufferToDisplay bufferTypeToDisplay,
    float motionVectorOffset, float motionVectorScale,
    optixu::NativeBlockBuffer2D<float4> outputBuffer,
    uint2 imageSize) {
    uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                   blockDim.y * blockIdx.y + threadIdx.y);
    if (launchIndex.x >= imageSize.x ||
        launchIndex.y >= imageSize.y)
        return;

    uint32_t linearIndex = launchIndex.y * imageSize.x + launchIndex.x;
    float4 value = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    switch (bufferTypeToDisplay) {
    case BufferToDisplay::NoisyBeauty:
    case BufferToDisplay::DenoisedBeauty: {
        auto typedLinearBuffer = reinterpret_cast<float4*>(linearBuffer);
        value = typedLinearBuffer[linearIndex];
        // simple tone-map
        value.x = 1 - std::exp(-value.x);
        value.y = 1 - std::exp(-value.y);
        value.z = 1 - std::exp(-value.z);
        break;
    }
    case BufferToDisplay::Albedo: {
        auto typedLinearBuffer = reinterpret_cast<float4*>(linearBuffer);
        value = typedLinearBuffer[linearIndex];
        break;
    }
    case BufferToDisplay::Normal: {
        auto typedLinearBuffer = reinterpret_cast<float4*>(linearBuffer);
        value = typedLinearBuffer[linearIndex];
        value.x = 0.5f + 0.5f * value.x;
        value.y = 0.5f + 0.5f * value.y;
        value.z = 0.5f + 0.5f * value.z;
        break;
    }
    case BufferToDisplay::Flow: {
        auto typedLinearBuffer = reinterpret_cast<float2*>(linearBuffer);
        float2 f2Value = typedLinearBuffer[linearIndex];
        value = make_float4(fminf(fmaxf(motionVectorScale * f2Value.x + motionVectorOffset, 0.0f), 1.0f),
                            fminf(fmaxf(motionVectorScale * f2Value.y + motionVectorOffset, 0.0f), 1.0f),
                            motionVectorOffset, 1.0f);
        break;
    }
    default:
        break;
    }
    outputBuffer.write(launchIndex, value);
}
