#include "hip/hip_runtime.h"
﻿#pragma once

#include "custom_primitive_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION static HitPointParameter get() {
        HitPointParameter ret;
        OptixPrimitiveType primType = optixGetPrimitiveType();
        // JP: 衝突したプリミティブのタイプを組み込み関数によって取得できる。
        // EN: The primitive type hit can be obtained using the intrinsic function.
        if (primType == OPTIX_PRIMITIVE_TYPE_TRIANGLE) {
            float2 bc = optixGetTriangleBarycentrics();
            ret.b1 = bc.x;
            ret.b2 = bc.y;
        }
        else if (primType == OPTIX_PRIMITIVE_TYPE_CUSTOM) {
            // JP: Intersection Programで設定したアトリビュート変数は optixu::getAttributes() で取得できる。
            // EN: Attribute variables set in the intersection program can be obtained using optixu::getAttributes().
            optixu::getAttributes<SphereAttributeSignature>(&ret.b1, &ret.b2);
        }
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryData geomData;

    CUDA_DEVICE_FUNCTION static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



// JP: レイとカスタムプリミティブとの衝突判定はIntersection Programで記述する。
// EN: Intersection program is used to describe the intersection between a ray vs a custom primitive.
CUDA_DEVICE_KERNEL void RT_IS_NAME(intersectSphere)() {
    auto sbtr = HitGroupSBTRecordData::get();
    const GeometryData &geom = sbtr.geomData;
    uint32_t primIndex = optixGetPrimitiveIndex();
    const SphereParameter &param = geom.paramBuffer[primIndex];
    const float3 rayOrg = optixGetObjectRayOrigin();
    const float3 rayDir = optixGetObjectRayDirection();

    float3 nDir = normalize(rayDir);

    float3 co = rayOrg - param.center;
    float b = dot(nDir, co);

    float D = b * b - (sqLength(co) - param.radius * param.radius);
    if (D < 0)
        return;

    float sqrtD = std::sqrt(D);
    float t0 = -b - sqrtD;
    float t1 = -b + sqrtD;
    bool isFront = t0 >= 0;
    float t = isFront ? t0 : t1;
    if (t < 0)
        return;

    float3 np = normalize(co + t * nDir);
    float theta = std::acos(std::fmin(std::fmax(np.z, -1.0f), 1.0f));
    float phi = std::fmod(std::atan2(np.y, np.x) + 2 * Pi, 2 * Pi);

    // JP: ペイロードと同様に、対応するreportIntersection()/getAttributes()で
    //     明示的にテンプレート引数を渡すことで型の不一致を検出できるようにすることを推奨する。
    // EN: It is recommended to explicitly pass template arguments to corresponding
    //     reportIntersection()/getAttributes() to detect type mismatch similar to payloads.
    optixu::reportIntersection<SphereAttributeSignature>(t, isFront ? 0 : 1, theta, phi);
}

CUDA_DEVICE_KERNEL void RT_RG_NAME(raygen)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float>(launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + 0.5f) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    float3 color;
    optixu::trace<PayloadSignature>(
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        color);

    plp.resultBuffer[launchIndex] = make_float4(color, 1.0f);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss)() {
    float3 color = make_float3(0, 0, 0.1f);
    optixu::setPayloads<PayloadSignature>(&color);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(closesthit)() {
    auto sbtr = HitGroupSBTRecordData::get();
    const GeometryData &geom = sbtr.geomData;
    auto hp = HitPointParameter::get();

    float3 sn;
    OptixPrimitiveType primType = optixGetPrimitiveType();
    if (primType == OPTIX_PRIMITIVE_TYPE_TRIANGLE) {
        const Triangle &triangle = geom.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geom.vertexBuffer[triangle.index0];
        const Vertex &v1 = geom.vertexBuffer[triangle.index1];
        const Vertex &v2 = geom.vertexBuffer[triangle.index2];

        float b0 = 1 - (hp.b1 + hp.b2);
        sn = b0 * v0.normal + hp.b1 * v1.normal + hp.b2 * v2.normal;
        sn = normalize(sn);
    }
    else if (primType == OPTIX_PRIMITIVE_TYPE_CUSTOM) {
        //const SphereParameter &param = geom.paramBuffer[hp.primIndex];
        float theta = hp.b1;
        float phi = hp.b2;
        float sinTheta = std::sin(theta);
        sn = make_float3(std::cos(phi) * sinTheta, std::sin(phi) * sinTheta, std::cos(theta));
    }

    sn = normalize(optixTransformNormalFromObjectToWorldSpace(sn));

    // JP: 法線を可視化。
    //     このサンプルでは単一のGASしか使っていないためオブジェクト空間からワールド空間への変換は無い。
    // EN: Visualize the normal.
    //     There is no object to world space transform since this sample uses only a single GAS.
    float3 color = 0.5f * sn + make_float3(0.5f);
    optixu::setPayloads<PayloadSignature>(&color);
}
