#include "hip/hip_runtime.h"
﻿#pragma once

#include "shared.h"

#define M_PI 3.14159265

namespace Sample {

using namespace Shared;

extern "C" __constant__ PipelineLaunchParameters plp;



// JP: このクラスのようにシステマティックにuint32_t&にせずに、
//     個別に適切なペイロードの渡し方を考えたほうが性能は良いかもしれない。
// EN: It is possibly better to individually tune how to pass a payload
//     unlike this class which systematically uses uint32_t &.
template <typename PayloadType>
union PayloadAccessor {
    PayloadType raw;
    uint32_t asUInt[(sizeof(PayloadType) + 3) / 4];
    static_assert(sizeof(PayloadType) <= 8 * 4, "sizeof(PayloadType) must be within 8 DWords.");

    RT_FUNCTION PayloadAccessor() {
        for (int i = 0; i < sizeof(asUInt) / 4; ++i)
            asUInt[i] = optixUndefinedValue();
    }

    RT_FUNCTION uint32_t &operator[](uint32_t idx) {
        return asUInt[idx];
    }

    RT_FUNCTION void getAll() {
        constexpr uint32_t numSlots = sizeof(asUInt) / 4;
        if (numSlots > 0)
            asUInt[0] = optixGetPayload_0();
        if (numSlots > 1)
            asUInt[1] = optixGetPayload_1();
        if (numSlots > 2)
            asUInt[2] = optixGetPayload_2();
        if (numSlots > 3)
            asUInt[3] = optixGetPayload_3();
        if (numSlots > 4)
            asUInt[4] = optixGetPayload_4();
        if (numSlots > 5)
            asUInt[5] = optixGetPayload_5();
        if (numSlots > 6)
            asUInt[6] = optixGetPayload_6();
        if (numSlots > 7)
            asUInt[7] = optixGetPayload_7();
    }

    RT_FUNCTION void setAll() const {
        constexpr uint32_t numSlots = sizeof(asUInt) / 4;
        if (numSlots > 0)
            optixSetPayload_0(asUInt[0]);
        if (numSlots > 1)
            optixSetPayload_1(asUInt[1]);
        if (numSlots > 2)
            optixSetPayload_2(asUInt[2]);
        if (numSlots > 3)
            optixSetPayload_3(asUInt[3]);
        if (numSlots > 4)
            optixSetPayload_4(asUInt[4]);
        if (numSlots > 5)
            optixSetPayload_5(asUInt[5]);
        if (numSlots > 6)
            optixSetPayload_6(asUInt[6]);
        if (numSlots > 7)
            optixSetPayload_7(asUInt[7]);
    }
};



struct Ray {
    float3 origin;
    float3 direction;
    float tmin;
    float tmax;
    float time;

    RT_FUNCTION static Ray getWorld() {
        Ray ret;
        ret.origin = optixGetWorldRayOrigin();
        ret.direction = optixGetWorldRayDirection();
        ret.tmin = optixGetRayTmin();
        ret.tmax = optixGetRayTmax();
        ret.time = optixGetRayTime();
        return ret;
    }
    RT_FUNCTION static Ray getObject() {
        Ray ret;
        ret.origin = optixGetObjectRayOrigin();
        ret.direction = optixGetObjectRayDirection();
        ret.tmin = optixGetRayTmin();
        ret.tmax = optixGetRayTmax();
        ret.time = optixGetRayTime();
        return ret;
    }
};

struct SearchRayPayload {
    PCG32RNG rng;
    float3 alpha;
    float3 contribution;
    float3 origin;
    float3 direction;
    struct {
        uint32_t pathLength;
        bool specularBounce : 1;
        bool terminate : 1;
    };
};

struct VisibilityRayPayload {
    float visibility;
};

struct HitPointParameter {
    float b0, b1;
    int32_t primIndex;

    RT_FUNCTION static HitPointParameter get() {
        HitPointParameter ret;
        if (optixIsTriangleHit()) {
            float2 bc = optixGetTriangleBarycentrics();
            ret.b0 = 1 - bc.x - bc.y;
            ret.b1 = bc.x;
        }
        else {
            ret.b0 = __uint_as_float(optixGetAttribute_0());
            ret.b1 = __uint_as_float(optixGetAttribute_1());
        }
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};



RT_PROGRAM void __raygen__pathtracing() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    int32_t index = plp.imageSize.x * launchIndex.y + launchIndex.x;

    PCG32RNG rng = plp.rngBuffer[index];

    float x = (float)(launchIndex.x + rng.getFloat0cTo1o()) / plp.imageSize.x;
    float y = (float)(launchIndex.y + rng.getFloat0cTo1o()) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    OptixTraversableHandle topGroup = plp.travHandles[plp.travIndex];

    PayloadAccessor<SearchRayPayload*> payloadPtr;
    SearchRayPayload payload;
    payload.alpha = make_float3(1.0f, 1.0f, 1.0f);
    payload.contribution = make_float3(0.0f, 0.0f, 0.0f);
    payload.rng = rng;
    payload.pathLength = 1;
    payload.terminate = false;
    payloadPtr.raw = &payload;
    while (true) {
        optixTrace(topGroup, origin, direction, 0.0f, INFINITY, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                   RayType_Search, NumRayTypes, RayType_Search,
                   payloadPtr[0], payloadPtr[1]);
        if (payload.terminate || payload.pathLength >= 10)
            break;

        origin = payload.origin;
        direction = payload.direction;
        ++payload.pathLength;
    }

    plp.rngBuffer[index] = payload.rng;
    float3 cumResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.numAccumFrames > 1) {
#if defined(USE_BUFFER2D)
        float4 cumResultF4 = plp.accumBuffer[launchIndex];
#else
        float4 cumResultF4 = plp.accumBuffer[index];
#endif
        cumResult = make_float3(cumResultF4.x, cumResultF4.y, cumResultF4.z);
    }
#if defined(USE_BUFFER2D)
    plp.accumBuffer.write(launchIndex, make_float4(cumResult + payload.contribution, 1.0f));
#else
    plp.accumBuffer[index] = make_float4(cumResult + payload.contribution, 1.0f);
#endif
}

RT_PROGRAM void __miss__searchRay() {
    PayloadAccessor<SearchRayPayload*> payloadPtr;
    payloadPtr.getAll();
    SearchRayPayload &payload = *payloadPtr.raw;

    payload.contribution = payload.contribution + payload.alpha * make_float3(0.01f, 0.01f, 0.01f);
    payload.terminate = true;

    //payload.setAll();
}

RT_CALLABLE_PROGRAM float3 __direct_callable__sampleTexture(uint32_t texID, float2 texCoord) {
    hipTextureObject_t texture = plp.textures[texID];
    float4 texValue = tex2D<float4>(texture, texCoord.x, texCoord.y);
    return make_float3(texValue.x, texValue.y, texValue.z);
}

RT_PROGRAM void __closesthit__shading_diffuse() {
    auto sbtr = optixu::getHitGroupSBTRecordData();
    auto matData = reinterpret_cast<const MaterialData*>(plp.materialData);
    auto geomInstData = reinterpret_cast<const GeometryData*>(plp.geomInstData);

    const MaterialData &mat = matData[sbtr.materialData];
    const GeometryData &geom = geomInstData[sbtr.geomInstData];

    OptixTraversableHandle topGroup = plp.travHandles[plp.travIndex];

    auto hitPointParam = HitPointParameter::get();

    PayloadAccessor<SearchRayPayload*> payloadPtr;
    payloadPtr.getAll();
    SearchRayPayload &payload = *payloadPtr.raw;

    PCG32RNG &rng = payload.rng;

    const Triangle &tri = geom.triangleBuffer[hitPointParam.primIndex];
    const Vertex &v0 = geom.vertexBuffer[tri.index0];
    const Vertex &v1 = geom.vertexBuffer[tri.index1];
    const Vertex &v2 = geom.vertexBuffer[tri.index2];
    float b0 = hitPointParam.b0;
    float b1 = hitPointParam.b1;
    float b2 = 1 - (b0 + b1);
    float3 p = optixTransformPointFromObjectToWorldSpace(b0 * v0.position + b1 * v1.position + b2 * v2.position);
    float3 sn = normalize(optixTransformNormalFromObjectToWorldSpace(b0 * v0.normal + b1 * v1.normal + b2 * v2.normal));

    float3 vOut = -optixGetWorldRayDirection();
    bool isFrontFace = dot(vOut, sn) > 0;
    if (!isFrontFace)
        sn = -sn;
    p = p + sn * 0.001f;

    //// Visualize normal
    //payload.contribution = 0.5f * sn + make_float3(0.5f, 0.5f, 0.5f);
    //payload.terminate = true;
    //return;

    float3 albedo = mat.albedo;
    if (mat.misc != 0xFFFFFFFF) {
        // Demonstrate how to use texture sampling and direct callable program.
        float2 texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;
        albedo = optixDirectCall<float3>(mat.program, mat.texID, texCoord);
    }

    const float3 LightRadiance = make_float3(20, 20, 20);
    // Hard-coded directly visible light
    if (sbtr.materialData == plp.matLightIndex &&
        isFrontFace &&
        (payload.pathLength == 1 || payload.specularBounce)) {
        payload.contribution = payload.contribution + payload.alpha * LightRadiance;
    }

    // Next Event Estimation
    {
        // Use hard-coded area light for simplicity.
        float3 lp = make_float3(-0.25f, 0.99f, -0.25f) +
            rng.getFloat0cTo1o() * make_float3(0.5f, 0, 0) +
            rng.getFloat0cTo1o() * make_float3(0, 0, 0.5f);
        float areaPDF = 4.0f;
        float3 lpn = make_float3(0, -1, 0);

        float3 shadowRayDir = lp - p;
        float dist2 = dot(shadowRayDir, shadowRayDir);
        float dist = std::sqrt(dist2);
        shadowRayDir = shadowRayDir / dist;
        float cosLight = dot(lpn, -shadowRayDir);
        float3 Le = cosLight > 0 ? LightRadiance : make_float3(0, 0, 0);

        PayloadAccessor<VisibilityRayPayload> shadowPayload;
        shadowPayload.raw.visibility = 1.0f;
        optixTrace(topGroup, p, shadowRayDir, 0.0f, dist * 0.999f, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                   RayType_Visibility, NumRayTypes, RayType_Visibility,
                   shadowPayload[0]);

        float cosSP = dot(sn, shadowRayDir);
        float G = shadowPayload.raw.visibility * std::fabs(cosSP) * std::fabs(cosLight) / dist2;
        float3 fs = cosSP > 0 ? albedo / M_PI : make_float3(0, 0, 0);
        float3 contribution = payload.alpha * fs * G * Le / areaPDF;
        payload.contribution = payload.contribution + contribution;
    }

    const auto makeCoordinateSystem = [](const float3 &n, float3* s, float3* t) {
        float sign = n.z >= 0 ? 1 : -1;
        float a = -1 / (sign + n.z);
        float b = n.x * n.y * a;
        *s = make_float3(1 + sign * n.x * n.x * a, sign * b, -sign * n.x);
        *t = make_float3(b, sign + n.y * n.y * a, -n.y);
    };

    float3 s;
    float3 t;
    makeCoordinateSystem(sn, &s, &t);

    // Sampling incoming direction.
    float phi = 2 * M_PI * rng.getFloat0cTo1o();
    float theta = std::asin(std::sqrt(rng.getFloat0cTo1o()));
    float sinTheta = std::sin(theta);
    float3 vIn = make_float3(std::cos(phi) * sinTheta, std::sin(phi) * sinTheta, std::cos(theta));
    vIn = make_float3(dot(make_float3(s.x, t.x, sn.x), vIn),
                      dot(make_float3(s.y, t.y, sn.y), vIn),
                      dot(make_float3(s.z, t.z, sn.z), vIn));
    payload.alpha = payload.alpha * albedo;
    payload.origin = p;
    payload.direction = vIn;
    payload.specularBounce = false;
    payload.terminate = false;

    //payload.setAll();
}

// JP: それなりの規模のパストレーシングを実装する場合はプログラムは基本的に共通化して
//     差異のある部分をCallable Programなどで呼び分けるほうが実用的だが、
//     ここではデモ目的であえて別のプログラムとする。
// EN: When implementing a moderately complex path tracing,
//     it appears better to basically use a common program and callable programs for different behaviors,
//     but here define another program on purpose for demonstration.
RT_PROGRAM void __closesthit__shading_specular() {
    auto sbtr = optixu::getHitGroupSBTRecordData();
    auto matData = reinterpret_cast<const MaterialData*>(plp.materialData);
    auto geomInstData = reinterpret_cast<const GeometryData*>(plp.geomInstData);

    const MaterialData &mat = matData[sbtr.materialData];
    const GeometryData &geom = geomInstData[sbtr.geomInstData];

    auto hitPointParam = HitPointParameter::get();

    PayloadAccessor<SearchRayPayload*> payloadPtr;
    payloadPtr.getAll();
    SearchRayPayload &payload = *payloadPtr.raw;

    PCG32RNG &rng = payload.rng;

    const Triangle &tri = geom.triangleBuffer[hitPointParam.primIndex];
    const Vertex &v0 = geom.vertexBuffer[tri.index0];
    const Vertex &v1 = geom.vertexBuffer[tri.index1];
    const Vertex &v2 = geom.vertexBuffer[tri.index2];
    float b0 = hitPointParam.b0;
    float b1 = hitPointParam.b1;
    float b2 = 1 - (b0 + b1);
    float3 p = optixTransformPointFromObjectToWorldSpace(b0 * v0.position + b1 * v1.position + b2 * v2.position);
    float3 sn = normalize(optixTransformNormalFromObjectToWorldSpace(b0 * v0.normal + b1 * v1.normal + b2 * v2.normal));
    p = p + sn * 0.001f;

    float3 albedo = mat.albedo;
    if (mat.misc != 0xFFFFFFFF) {
        // Demonstrate how to use texture sampling and direct callable program.
        float2 texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;
        albedo = optixDirectCall<float3>(mat.program, mat.texID, texCoord);
    }

    float3 vOut = -optixGetWorldRayDirection();

    // Sampling incoming direction (delta distribution).
    float3 vIn = normalize(2 * dot(vOut, sn) * sn - vOut);
    payload.alpha = payload.alpha * albedo;
    payload.origin = p;
    payload.direction = vIn;
    payload.specularBounce = true;
    payload.terminate = false;

    //payload.setAll();
}

RT_PROGRAM void __anyhit__visibility() {
    PayloadAccessor<VisibilityRayPayload> payload;

    payload.raw.visibility = 0.0f;
    payload.setAll();

    optixTerminateRay();
}

RT_PROGRAM void __exception__print() {
    uint3 launchIndex = optixGetLaunchIndex();
    int32_t code = optixGetExceptionCode();
    printf("(%u, %u, %u): Exception: %u\n", launchIndex.x, launchIndex.y, launchIndex.z, code);
}

}
